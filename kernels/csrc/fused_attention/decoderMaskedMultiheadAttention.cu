#include "hip/hip_runtime.h"
// Inspired by TRT-LLM.
// Modified by Haotian Tang and Shang Yang.
// @article{lin2024qserve,
//   title={QServe: W4A8KV4 Quantization and System Co-design for Efficient LLM Serving},
//   author={Lin*, Yujun and Tang*, Haotian and Yang*, Shang and Zhang, Zhekai and Xiao, Guangxuan and Gan, Chuang and Han, Song},
//   journal={arXiv preprint arXiv:2405.04532},
//   year={2024}
// }
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "decoderMaskedMultiheadAttention.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <type_traits>
#include "decoderMaskedMultiheadAttentionTemplate.hpp"

namespace mmha
{
////////////////////////////////////////////////////////////////////////////////////////////////////


// Forward declaration of the kernel launcher to avoid including decoderMaskedMultiheadAttentionLaunch.h
template <typename T, int Dh>
inline size_t multi_block_grid_setup(const Multihead_attention_params<T>& params,
    int threads_per_block, int tlength, bool do_multi_block)
{
    if (!do_multi_block)
    {
        return 1;
    }

    auto constexpr threads_per_value = mmha::threads_per_value<T>(mmha::dh_max(Dh));

    // Make sure: seq_len_tile * threads_per_value <= threads_per_block (for multi_block_mode)
    params.seq_len_tile = std::floor(threads_per_block / threads_per_value);

    assert(params.seq_len_tile <= params.max_seq_len_tile);

    params.timesteps_per_block = mmha::divUp(tlength, params.seq_len_tile);

#ifndef ENABLE_MULTI_BLOCK_OPTION
    do_multi_block = false;
#endif

    // Return the sequence length tile if using multi block modes.
    return params.seq_len_tile;
}


#define MMHA_LAUNCH_CHECK(DYNAMIC_THDS_PER_BLOCK)                                                                      \
    std::size_t const dynamic_smem_sz{                                                                                 \
        mmha::smem_size_in_bytes<T, Dh, DO_MULTI_BLOCK>(params, DYNAMIC_THDS_PER_BLOCK)};                              \
    /* Set 46KB threshold here because we have to take static/driver shared memory into consideration. */              \
    if (dynamic_smem_sz >= 46 * 1024)                                                                                  \
    {                                                                                                                  \
        hipError_t res = hipFuncSetAttribute(reinterpret_cast<const void*>(mmha::masked_multihead_attention_kernel<T), T_cache, KVCacheBuffer, Dh,  \
                                                   DYNAMIC_THDS_PER_BLOCK, DO_MULTI_BLOCK, INT4KV, KV_WITH_ZEROS, SMEM_PRELOAD>,                 \
            hipFuncAttributeMaxDynamicSharedMemorySize, dynamic_smem_sz);                                             \
    }                                                                                                                  \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&available_blocks,                                                   \
        mmha::masked_multihead_attention_kernel<T, T_cache, KVCacheBuffer, Dh, DYNAMIC_THDS_PER_BLOCK,      \
            DO_MULTI_BLOCK, INT4KV, KV_WITH_ZEROS, SMEM_PRELOAD>,                                                                                           \
        DYNAMIC_THDS_PER_BLOCK, dynamic_smem_sz);


#define MMHA_KERNEL(DYNAMIC_THDS_PER_BLOCK)                                                                            \
    std::size_t const dynamic_smem_sz{                                                                                 \
        mmha::smem_size_in_bytes<T, Dh, DO_MULTI_BLOCK>(params, DYNAMIC_THDS_PER_BLOCK)};                              \
    /* Set 46KB threshold here because we have to take static/driver shared memory into consideration. */              \
    if (dynamic_smem_sz >= 46 * 1024)                                                                                  \
    {                                                                                                                  \
        hipError_t res = hipFuncSetAttribute(reinterpret_cast<const void*>(                                                                        \
            mmha::masked_multihead_attention_kernel<T), T_cache, KVCacheBuffer, Dh, DYNAMIC_THDS_PER_BLOCK,             \
                 DO_MULTI_BLOCK, INT4KV, KV_WITH_ZEROS, SMEM_PRELOAD>,                                      \
            hipFuncAttributeMaxDynamicSharedMemorySize, dynamic_smem_sz);                                             \
    }                                                                                                                  \
    mmha::masked_multihead_attention_kernel<T, T_cache, KVCacheBuffer, Dh, DYNAMIC_THDS_PER_BLOCK,                     \
         DO_MULTI_BLOCK, INT4KV, KV_WITH_ZEROS, SMEM_PRELOAD>                                               \
        <<<grid, DYNAMIC_THDS_PER_BLOCK, dynamic_smem_sz, stream>>>(params, kv_cache_buffer);


// if resources are not enough to launch 512 threads per block, we will fallback to 256.
#define MMHA_LAUNCH_512_BLOCKSIZE()                                                                                    \
    int available_blocks = -1;                                                                                         \
    MMHA_LAUNCH_CHECK(512);                                                                                            \
    if (available_blocks <= 0)                                                                                         \
    {                                                                                                                  \
        MMHA_KERNEL(256);                                                                                              \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        MMHA_KERNEL(512);                                                                                              \
    }

// if resources are not enough to launch 1024 threads per block, we will fallback to 512.
#define MMHA_LAUNCH_1024_BLOCKSIZE()                                                                                   \
    int available_blocks = -1;                                                                                         \
    MMHA_LAUNCH_CHECK(1024);                                                                                           \
    if (available_blocks <= 0)                                                                                         \
    {                                                                                                                  \
        MMHA_LAUNCH_512_BLOCKSIZE();                                                                                   \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        MMHA_KERNEL(1024);                                                                                             \
    }

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, typename T_cache, typename KVCacheBuffer, typename KernelParamsType, int Dh, int THDS_PER_BLOCK,
    bool DO_MULTI_BLOCK, bool INT4KV, bool KV_WITH_ZEROS, bool SMEM_PRELOAD>
void mmha_launch_kernel_ex(
    const KernelParamsType& params, const KVCacheBuffer& kv_cache_buffer, const hipStream_t& stream, int tlength)
{
    std::size_t const seq_len_tile{mmha::multi_block_grid_setup<T, Dh>(
        params, THDS_PER_BLOCK, tlength, DO_MULTI_BLOCK)};
    dim3 grid{static_cast<unsigned>(params.num_heads), static_cast<unsigned>(params.batch_size),
        static_cast<unsigned>(seq_len_tile)};

    if (DO_MULTI_BLOCK)
    {
        MMHA_KERNEL(THDS_PER_BLOCK);
    }
    else
    {
        const int kernel_total_blocks = params.batch_size * params.num_heads;
        // Don't tune the block size if batchxhead is large enough.
        // The max number of warps we can launch per SM is 32 limited by registers.
        if (kernel_total_blocks >= params.multi_processor_count * 4)
        {
            MMHA_KERNEL(THDS_PER_BLOCK);
            return;
        }

        // Tune block size based on batchxhead to increase occupancy.
        int num_blocks_per_sm = -1;
        // Set 0 dynamic shared memory size as we need the number of available blocks limited by registers.
        // Dynamic shared memory is fixed for different block size.
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm,
            mmha::masked_multihead_attention_kernel<T, T_cache, KVCacheBuffer, Dh, THDS_PER_BLOCK,
                DO_MULTI_BLOCK, INT4KV, KV_WITH_ZEROS, SMEM_PRELOAD>,
            THDS_PER_BLOCK, 0);

        int block_size_factor = min(
            mmha::divUp(params.multi_processor_count * num_blocks_per_sm, kernel_total_blocks), num_blocks_per_sm);
        // Max block size is 1024.
        const int dynamic_block_size = min(THDS_PER_BLOCK * block_size_factor, 1024);

        // Make sure number of threads per block is power of 2.
        if (dynamic_block_size <= 256)
        {
            MMHA_KERNEL(256);
        }
        else if (dynamic_block_size <= 512)
        {
            // Check if the kernel with new block size can be launched in terms of resources.
            MMHA_LAUNCH_512_BLOCKSIZE();
        }
        else if (dynamic_block_size <= 1024)
        {
            // Check if the kernel with new block size can be launched in terms of resources.
            MMHA_LAUNCH_1024_BLOCKSIZE();
        }
    }
}

template <typename T, typename KVCacheBuffer, typename KernelParamsType, int Dh, int THDS_PER_BLOCK,
    bool DO_MULTI_BLOCK>
void mmha_launch_kernel_dispatch_4bits_kv_cache(
    const KernelParamsType& params, const KVCacheBuffer& kv_cache_buffer, const hipStream_t& stream, int tlength)
{
    if (params.int4_kv_cache)
    {
        if (params.kv_cache_with_zeros)
        {
            if (params.timestep < 2048)
            {
                // Note: the 4bit kv_cache is still packed in int8_t.
                mmha_launch_kernel_ex<T, int8_t, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK,
                    DO_MULTI_BLOCK, true, true, true>(params, kv_cache_buffer, stream, tlength);
            }
            else
            {
                // Note: the 4bit kv_cache is still packed in int8_t.
                mmha_launch_kernel_ex<T, int8_t, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK,
                    DO_MULTI_BLOCK, true, true, false>(params, kv_cache_buffer, stream, tlength);
            }
        }
        else
        {
            if (params.timestep < 2048)
            {
                // Note: the 4bit kv_cache is still packed in int8_t.
                mmha_launch_kernel_ex<T, int8_t, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK,
                    DO_MULTI_BLOCK, true, false, true>(params, kv_cache_buffer, stream, tlength);
            }
            else
            {
                // Note: the 4bit kv_cache is still packed in int8_t.
                mmha_launch_kernel_ex<T, int8_t, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK,
                    DO_MULTI_BLOCK, true, false, false>(params, kv_cache_buffer, stream, tlength);
            }

        }
    }
    else
    {
        // this should never happen
        // mmha_launch_kernel_ex<T, T, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK, DO_MULTI_BLOCK, true>(
        //     params, kv_cache_buffer, stream, tlength);
    }
}

template <typename T, typename KVCacheBuffer, typename KernelParamsType, int Dh, int THDS_PER_BLOCK,
    bool DO_MULTI_BLOCK>
void mmha_launch_kernel_dispatch_8bits_kv_cache(
    const KernelParamsType& params, const KVCacheBuffer& kv_cache_buffer, const hipStream_t& stream, int tlength)
{
    if (params.int8_kv_cache)
    {
        if (params.kv_cache_with_zeros)
        {
            mmha_launch_kernel_ex<T, int8_t, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK,
                DO_MULTI_BLOCK, false, true, false>(params, kv_cache_buffer, stream, tlength);
        }
        else
        {
            mmha_launch_kernel_ex<T, int8_t, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK,
                DO_MULTI_BLOCK, false, false, false>(params, kv_cache_buffer, stream, tlength);
        }
    }
#ifdef ENABLE_FP8
    else if (params.fp8_kv_cache)
    {
        mmha_launch_kernel_ex<T, __hip_fp8_e4m3_fnuz, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK,
            DO_MULTI_BLOCK, false, false, false>(params, kv_cache_buffer, stream, tlength);
    }
#endif // ENABLE_FP8
    else
    {
        mmha_launch_kernel_ex<T, T, KVCacheBuffer, KernelParamsType, Dh, THDS_PER_BLOCK, DO_MULTI_BLOCK, false, false, false>(
            params, kv_cache_buffer, stream, tlength);
    }
}

template <typename T, typename KVCacheBuffer, typename KernelParamsType, int Dh>
void mmha_launch_kernel_dispatch(
    const KernelParamsType& params, const KVCacheBuffer& kv_cache_buffer, const hipStream_t& stream)
{
    int const tlength = params.timestep;
    bool int4_kv_cache = params.int4_kv_cache;
    if (int4_kv_cache)
    {
        if (tlength < 1024)
        {
            mmha_launch_kernel_dispatch_4bits_kv_cache<T, KVCacheBuffer, KernelParamsType, Dh, 256, false>(
                params, kv_cache_buffer, stream, tlength);
        }
        else
        {
            if (params.multi_block_mode)
            {
                mmha_launch_kernel_dispatch_4bits_kv_cache<T, KVCacheBuffer, KernelParamsType, Dh, 256, true>(
                    params, kv_cache_buffer, stream, tlength);
            }
            else
            {
                mmha_launch_kernel_dispatch_4bits_kv_cache<T, KVCacheBuffer, KernelParamsType, Dh, 256, false>(
                    params, kv_cache_buffer, stream, tlength);
            }
        }
    }
    else    // int8_kv_cache
    {
        if (tlength < 1024)
        {
            mmha_launch_kernel_dispatch_8bits_kv_cache<T, KVCacheBuffer, KernelParamsType, Dh, 256, false>(
                params, kv_cache_buffer, stream, tlength);
        }
        else
        {
            if (params.multi_block_mode)
            {
                mmha_launch_kernel_dispatch_8bits_kv_cache<T, KVCacheBuffer, KernelParamsType, Dh, 256, true>(
                    params, kv_cache_buffer, stream, tlength);
            }
            else
            {
                mmha_launch_kernel_dispatch_8bits_kv_cache<T, KVCacheBuffer, KernelParamsType, Dh, 256, false>(
                    params, kv_cache_buffer, stream, tlength);
            }
        }
    }
}

template <typename T, typename KVCacheBuffer, typename KernelParamsType, int Dh>
void mmha_launch_kernel(
    const KernelParamsType& params, const KVCacheBuffer& kv_cache_buffer, const hipStream_t& stream)
{
    // assert((params.rotary_embedding_dim != 0)
    //     == (params.position_embedding_type == PositionEmbeddingType::kROPE_GPT_NEOX
    //         || params.position_embedding_type == PositionEmbeddingType::kROPE_GPTJ));
    // if (params.beam_width == 1)
    // {
    //     mmha_launch_kernel_dispatch<T, KVCacheBuffer, KernelParamsType, Dh, false>(params, kv_cache_buffer, stream);
    // }
    // else
    // {
    //     mmha_launch_kernel_dispatch<T, KVCacheBuffer, KernelParamsType, Dh, true>(params, kv_cache_buffer, stream);
    // }
    mmha_launch_kernel_dispatch<T, KVCacheBuffer, KernelParamsType, Dh>(params, kv_cache_buffer, stream);
}

} // namespace mmha

namespace
{

#define MMHA_LAUNCH_KERNEL(Dh)                                                                                         \
    mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, Dh>(params, kv_cache_buffer, stream);               \
    break;

template <typename T, typename KVCacheBuffer, typename KERNEL_PARAMS_TYPE>
void multihead_attention_(
    const KERNEL_PARAMS_TYPE& params, const KVCacheBuffer& kv_cache_buffer, const hipStream_t& stream)
{
    switch (params.hidden_size_per_head)
    {
    // case 32: mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 32>(params, kv_cache_buffer, stream); break;
    // case 48: mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 48>(params, kv_cache_buffer, stream); break;
    // case 64: mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 64>(params, kv_cache_buffer, stream); break;
    // case 80: mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 80>(params, kv_cache_buffer, stream); break;
    // case 96: mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 96>(params, kv_cache_buffer, stream); break;
    // case 112:
    //     mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 112>(params, kv_cache_buffer, stream);
    //     break;
    case 128:
        mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 128>(params, kv_cache_buffer, stream);
        break;
    // case 144:
    //     mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 144>(params, kv_cache_buffer, stream);
    //     break;
    // case 160:
    //     mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 160>(params, kv_cache_buffer, stream);
    //     break;
    // case 192:
    //     mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 192>(params, kv_cache_buffer, stream);
    //     break;
    // case 224:
    //     mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 224>(params, kv_cache_buffer, stream);
    //     break;
    // case 256:
    //     mmha::mmha_launch_kernel<T, KVCacheBuffer, KERNEL_PARAMS_TYPE, 256>(params, kv_cache_buffer, stream);
    //     break;
    default: assert(false);
    }
}

#undef MMHA_LAUNCH_KERNEL

} // namespace

////////////////////////////////////////////////////////////////////////////////////////////////////

#define INSTANTIATE_MMHA_NORMAL_AND_PAGED(T)                                                          \
    void masked_multihead_attention(const Multihead_attention_params<T>& params,                      \
        const KVBlockArray& kv_cache_buffer, const hipStream_t& stream)                                               \
    {                                                                                                                  \
        multihead_attention_<T, KVBlockArray, Multihead_attention_params<T>>(                         \
            params, kv_cache_buffer, stream);                                                                          \
    }                                                                                                                  \
    void masked_multihead_attention(const Multihead_attention_params<T>& params,                      \
        const KVLinearBuffer& kv_cache_buffer, const hipStream_t& stream)                                             \
    {                                                                                                                  \
        multihead_attention_<T, KVLinearBuffer, Multihead_attention_params<T>>(                       \
            params, kv_cache_buffer, stream);                                                                          \
    }
//INSTANTIATE_MMHA_NORMAL_AND_PAGED(float, true)
// INSTANTIATE_MMHA_NORMAL_AND_PAGED(float)
//INSTANTIATE_MMHA_NORMAL_AND_PAGED(uint16_t, true)
INSTANTIATE_MMHA_NORMAL_AND_PAGED(uint16_t)
#undef INSTANTIATE_MMHA_NORMAL_AND_PAGED

////////////////////////////////////////////////////////////////////////////////////////////////////

